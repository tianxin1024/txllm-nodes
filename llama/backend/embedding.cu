#include "hip/hip_runtime.h"
#include "backend/embedding.h"
#include "backend/utils.h"

namespace nn {

class RawEmbedding::impl {
public:
    class NormalImpl;
    class ParallelImpl;
    // class RowParallelImpl;
    float logit_scale{1.}; // For Cohere model
    virtual ~impl() = default;

    virtual core::Tensor &get_weight() = 0;

}; // end of class RawEmbedding

class RawEmbedding::impl::NormalImpl : public RawEmbedding::impl {
public:
    core::Tensor weight;
    unsigned int dim_model;
    core::DataType dtype;
    unsigned int begin;
    unsigned int end;
    float scale_factor;
    NormalImpl(const core::Context &ctx,
               unsigned int vocab_size,
               unsigned int dim_model,
               bool scale_weights,
               core::DataType dtype) :
        weight(ctx.parameter({vocab_size, dim_model}, dtype)),
        dim_model(dim_model),
        dtype(dtype),
        begin(0),
        end(vocab_size),
        scale_factor(scale_weights ? 1.0 / sqrtf(dim_model) : 1.0) {
    }

    core::Tensor &get_weight() {
        return weight;
    }

}; // end of class RawEmbedding::impl::NormalImpl

// class RawEmbedding::impl::RowParallelImpl : public RawEmbedding::impl {
// }; // end of class RawEmbedding::impl::RowParallelImpl

RawEmbedding::RawEmbedding(const core::Context &ctx,
                           int dim_model,
                           int vocab_size,
                           bool scale_weights,
                           core::DataType dtype,
                           bool parallel) :
    core::Layer() {
    int row_parallel = utils::get_int_env("CPM_EMB_ROW_PAR", 1);
    if (parallel) {
        // pimpl.reset(new impl::RowParallelImpl(ctx, vocab_size, dim_model, scale_weights, dtype));
    } else {
        pimpl.reset(new impl::NormalImpl(ctx, vocab_size, dim_model, scale_weights, dtype));
    }
    add_parameter("weight", pimpl->get_weight());
}

RawEmbedding::~RawEmbedding() = default;

} // namespace nn
